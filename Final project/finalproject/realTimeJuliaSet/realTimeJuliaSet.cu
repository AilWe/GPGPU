#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"

PFNGLBINDBUFFERARBPROC    glBindBuffer     = NULL;
PFNGLDELETEBUFFERSARBPROC glDeleteBuffers  = NULL;
PFNGLGENBUFFERSARBPROC    glGenBuffers     = NULL;
PFNGLBUFFERDATAARBPROC    glBufferData     = NULL;

#define     DIM    512
#define MY_PI 3.1415926

static float rotatef = 296.0;
static int updateTime = 1000;
static float updateAngle = 0.5;

struct hipComplex{
  float r;
  float i;
  __host__ __device__ hipComplex(float a, float b) : r(a), i(b) {}
  __device__ float magnitude2(void) {
    return r * r + i * i;
  }
  __device__ hipComplex operator*(const hipComplex& a){
    return hipComplex(r*a.r-i*a.i, i*a.r+r*a.i);
  }
  __device__ hipComplex operator+(const hipComplex& a){
    return hipComplex(r+a.r, i+a.i);
  }
};

__device__ int julia(int x, int y, float angle) {
  const float scale = 1.5;
  float jx = scale * (float)(DIM/2 - x)/(DIM/2);
  float jy = scale * (float)(DIM/2 - y)/(DIM/2);

  float factor = 0.578;
  hipComplex c(factor * cosf(angle), factor * sinf(angle));
//  hipComplex c(-0.8, angle / (2*MY_PI));
//  hipComplex c(-0.8, 0.156);
  hipComplex a(jx, jy);

  int i = 0;
  for (i = 0; i < 200; i++) {
    a = a * a + c;
    if (a.magnitude2() > 1000)
      return 0;
  }
  return i;
}

GLuint  bufferObj;
hipGraphicsResource *resource;

// based on ripple code, but uses uchar4 which is the type of data
// graphic inter op uses. see screenshot - basic2.png
__global__ void kernel( uchar4 *ptr , float angle) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia(x, y, angle/180.0*MY_PI);
    // accessing uchar4 vs unsigned char*
    ptr[offset].x = 255 * juliaValue;
    ptr[offset].y = 55 * juliaValue;
    ptr[offset].z = 25 * juliaValue;
    ptr[offset].w = 255;
}

static void key_func( unsigned char key, int x, int y ) {
    switch (key) {
        case 27:
            // clean up OpenGL and CUDA
            HANDLE_ERROR( hipGraphicsUnregisterResource( resource ) );
            glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 );
            glDeleteBuffers( 1, &bufferObj );
            exit(0);
    }
}

static void draw_func( void ) {
    // we pass zero as the last parameter, because out bufferObj is now
    // the source, and the field switches from being a pointer to a
    // bitmap to now mean an offset into a bitmap object
    HANDLE_ERROR( 
        hipGraphicsGLRegisterBuffer( &resource, 
                                      bufferObj, 
                                      cudaGraphicsMapFlagsNone ) );
    HANDLE_ERROR( hipGraphicsMapResources( 1, &resource, NULL ) );
    uchar4* devPtr;
    size_t  size;
    HANDLE_ERROR( 
        hipGraphicsResourceGetMappedPointer( (void**)&devPtr, 
                                              &size, 
                                              resource) );
    dim3    grids(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<grids,threads>>>( devPtr , rotatef);
    glDrawPixels( DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0 );
    glutSwapBuffers();
}

void update(int value){
  rotatef += updateAngle;
  printf ("rotatef: %f\n", rotatef);
  if (rotatef > 360.f)
    rotatef -= 360;
  glutPostRedisplay();
  glutTimerFunc(updateTime, update, 0);
}

int main( int argc, char **argv ) {
    hipDeviceProp_t  prop;
    int dev;

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );

    // tell CUDA which dev we will be using for graphic interop
    // from the programming guide:  Interoperability with OpenGL
    //     requires that the CUDA device be specified by
    //     cudaGLSetGLDevice() before any other runtime calls.

    HANDLE_ERROR( cudaGLSetGLDevice( dev ) );

    // these GLUT calls need to be made before the other OpenGL
    // calls, else we get a seg fault
    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
    glutInitWindowSize( DIM, DIM );
    glutCreateWindow( "bitmap" );

    glBindBuffer    = (PFNGLBINDBUFFERARBPROC)GET_PROC_ADDRESS("glBindBuffer");
    glDeleteBuffers = (PFNGLDELETEBUFFERSARBPROC)GET_PROC_ADDRESS("glDeleteBuffers");
    glGenBuffers    = (PFNGLGENBUFFERSARBPROC)GET_PROC_ADDRESS("glGenBuffers");
    glBufferData    = (PFNGLBUFFERDATAARBPROC)GET_PROC_ADDRESS("glBufferData");

    // the first three are standard OpenGL, the 4th is the CUDA reg 
    // of the bitmap these calls exist starting in OpenGL 1.5
    glGenBuffers( 1, &bufferObj );
    glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj );
    glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, DIM * DIM * 4, NULL, GL_DYNAMIC_DRAW_ARB );

//    HANDLE_ERROR( 
//        hipGraphicsGLRegisterBuffer( &resource, 
//                                      bufferObj, 
//                                      cudaGraphicsMapFlagsNone ) );

    // do work with the memory dst being on the GPU, gotten via mapping
//    HANDLE_ERROR( hipGraphicsMapResources( 1, &resource, NULL ) );
//    uchar4* devPtr;
//    size_t  size;
//    HANDLE_ERROR( 
//        hipGraphicsResourceGetMappedPointer( (void**)&devPtr, 
//                                              &size, 
//                                              resource) );

//    dim3    grids(DIM/16,DIM/16);
//    dim3    threads(16,16);
//    kernel<<<grids,threads>>>( devPtr , rotatef);
//    HANDLE_ERROR( hipGraphicsUnmapResources( 1, &resource, NULL ) );

    // set up GLUT and kick off main loop
    glutKeyboardFunc( key_func );
    glutDisplayFunc( draw_func );
    glutTimerFunc(updateTime, update, 0);
    glutMainLoop();
}
